#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "potential.h"
// for cuda profiler
#include "hip/hip_runtime_api.h"

//PowerSphericalPotential
//2  arguments: amp, alpha
double KeplerPotentialEval(double R,double Z, double phi, double t, struct potentialArg * potentialArgs){
    double * args= potentialArgs->args;
    //Get args
    double amp= *args++;
    double alpha= *args;
    //Calculate Rforce
    if ( alpha == 2. )
        return 0.5 * amp * log ( R*R+Z*Z);
    else
        return - amp * pow(R*R+Z*Z,1.-0.5*alpha) / (alpha - 2.);
}
double PowerSphericalPotentialRforce(double R,double Z, double phi,
				      double t,
				     struct potentialArg * potentialArgs){
  double * args= potentialArgs->args;
  //Get args
  double amp= *args++;
  double alpha= *args;
  //Calculate Rforce
  return - amp * R * pow(R*R+Z*Z,-0.5*alpha);
}
double PowerSphericalPotentialPlanarRforce(double R,double phi,
					   double t,
					   struct potentialArg * potentialArgs){
  double * args= potentialArgs->args;
  //Get args
  double amp= *args++;
  double alpha= *args;
  //Calculate Rforce
  return - amp * pow(R,-alpha + 1.);
}
double PowerSphericalPotentialzforce(double R,double Z,double phi,
				     double tt,
				     struct potentialArg * potentialArgs){
  double * args= potentialArgs->args;
  //Get args
  double amp= *args++;
  double alpha= *args;
  //Calculate zforce
  return - amp * Z * pow(R*R+Z*Z,-0.5*alpha);
}
double PowerSphericalPotentialPlanarR2deriv(double R,double phi,
					     double t,
					    struct potentialArg * potentialArgs){
  double * args= potentialArgs->args;
  //Get args
  double amp= *args++;
  double alpha= *args;
  //Calculate R2deriv
  return amp * (1. - alpha ) * pow(R,-alpha);
}
